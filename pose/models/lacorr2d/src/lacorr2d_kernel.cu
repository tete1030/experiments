#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void lacorr2d_forward_cuda_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* output,
    const int kernel_height,
    const int kernel_width,
    const int stride_height,
    const int stride_width,
    const int n_corr_h,
    const int n_corr_w,
    const int channel_size,
    const int height,
    const int width,
    const int64_t state_size) {
        const int64_t cudakernel_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (cudakernel_id < state_size) {
            const int corr_size = kernel_height * kernel_width;
            // ith sample in batch
            const int i_samp = blockIdx.z;
            // number of corr
            const int n_corr = n_corr_h * n_corr_w;
            // number of total pixels of corrs in each channel
            const int all_corr_size =  n_corr * corr_size;
            // ith channel
            const int i_channel = cudakernel_id / all_corr_size;
            // ith pixel in each channel
            const int i_all_corr = cudakernel_id % all_corr_size;
            // ith corr map in each channel
            const int i_corr = i_all_corr / corr_size;
            const int i_corr_h = i_corr / n_corr_w;
            const int i_corr_w = i_corr % n_corr_w;
            // left and top conner of current corr in input image
            const int left = stride_width * i_corr_w;
            const int top = stride_height * i_corr_h;

            // ith flatten pixel in each corr
            const int pos_kernel = i_all_corr % corr_size;
            // location in output
            const int y_out = pos_kernel / kernel_width + top;
            const int x_out = pos_kernel % kernel_width + left;
            // location in input for kernel use
            const int y_inp_k = blockIdx.y / kernel_width + top;
            const int x_inp_k = blockIdx.y % kernel_width + left;

            // location in input for multiplicand of kernel
            // (*_out - kernel_* / 2) : left/top conner of kernel projected on the input
            // (*_inp_k - top/left) : x/y inside kernel
            const int y_inp_bg = y_out - kernel_height / 2 + y_inp_k - top;
            const int x_inp_bg = x_out - kernel_width / 2 + x_inp_k - left;

            // pad 0 for multiplicand of kernel
            if (y_inp_bg < 0 || y_inp_bg >= height || x_inp_bg < 0 || x_inp_bg >= width) {
                return;
            }

            const int64_t index_out = i_samp * state_size + cudakernel_id;
            const int64_t index_bg = i_samp * channel_size * height * width + i_channel * height * width + y_inp_bg * width + x_inp_bg;
            const int64_t index_k = i_samp * channel_size * height * width + i_channel * height * width + y_inp_k * width + x_inp_k;
            auto out = output + index_out;
            auto bg_val = *(input + index_bg);
            auto k_val = *(input + index_k);
            
            atomicAdd(out, bg_val * k_val);
        }
}

std::vector<at::Tensor> lacorr2d_forward_cuda(
    at::Tensor input,
    int kernel_height,
    int kernel_width,
    int stride_height,
    int stride_width) {
    const int batch_size = input.size(0);
    const int channel_size = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    AT_ASSERT(kernel_width <= width, "kernel_width must be lesser than or equal to width")
    AT_ASSERT(kernel_height <= height, "kernel_height must be lesser than or equal to height")
    AT_ASSERT(stride_width <= width, "stride_width must be lesser than or equal to width")
    AT_ASSERT(stride_height <= height, "stride_height must be lesser than or equal to height")

    const int n_corr_w = (width - kernel_width) / stride_width + 1;
    const int n_corr_h = (height - kernel_height) / stride_height + 1;
    const int n_corr = n_corr_w * n_corr_h;
    const int corr_size = kernel_height * kernel_width;

    // working on pytorch 0.4.0 , have been changed in master 07/20/2018
    auto output = at::zeros(input.type(), std::vector<int64_t>{batch_size, channel_size, n_corr_h, n_corr_w, kernel_height, kernel_width});
    const int64_t state_size = (int64_t)channel_size * n_corr * corr_size;

    const int threads = 1024;
    const dim3 blocks((state_size + threads - 1) / threads, corr_size, batch_size);
    AT_DISPATCH_FLOATING_TYPES(input.type(), "lacorr2d_forward_cuda", ([&] {
        lacorr2d_forward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            input.data<scalar_t>(),
            output.data<scalar_t>(),
            kernel_height,
            kernel_width,
            stride_height,
            stride_width,
            n_corr_h,
            n_corr_w,
            channel_size,
            height,
            width,
            state_size);
    }));
    return {output};
}

template <typename scalar_t>
__global__ void lacorr2d_backward_cuda_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ grad_output,
    scalar_t* grad_input,
    const int kernel_height,
    const int kernel_width,
    const int stride_height,
    const int stride_width,
    const int n_corr_h,
    const int n_corr_w,
    const int channel_size,
    const int height,
    const int width,
    const int64_t state_size) {
        const int64_t cudakernel_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (cudakernel_id < state_size) {
            const int corr_size = kernel_height * kernel_width;
            // ith sample in batch
            const int i_samp = blockIdx.z;
            // number of corr
            const int n_corr = n_corr_h * n_corr_w;
            // number of total pixels of corrs in each channel
            const int all_corr_size =  n_corr * corr_size;
            // ith channel
            const int i_channel = cudakernel_id / all_corr_size;
            // ith pixel in each channel
            const int i_all_corr = cudakernel_id % all_corr_size;
            // ith corr map in each channel
            const int i_corr = i_all_corr / corr_size;
            const int i_corr_h = i_corr / n_corr_w;
            const int i_corr_w = i_corr % n_corr_w;
            // left and top conner of current corr in input image
            const int left = stride_width * i_corr_w;
            const int top = stride_height * i_corr_h;

            // ith flatten pixel in each corr
            const int pos_kernel = i_all_corr % corr_size;
            // location in output
            const int y_out = pos_kernel / kernel_width + top;
            const int x_out = pos_kernel % kernel_width + left;
            // location in input for kernel use
            const int y_inp_k = blockIdx.y / kernel_width + top;
            const int x_inp_k = blockIdx.y % kernel_width + left;

            // location in input for multiplicand of kernel
            // (*_out - kernel_* / 2) : left/top conner of kernel projected on the input
            // (*_inp_k - top/left) : x/y inside kernel
            const int y_inp_bg = y_out - kernel_height / 2 + y_inp_k - top;
            const int x_inp_bg = x_out - kernel_width / 2 + x_inp_k - left;

            // pad 0 for multiplicand of kernel
            if (y_inp_bg < 0 || y_inp_bg >= height || x_inp_bg < 0 || x_inp_bg >= width) {
                return;
            }

            const int64_t index_out = i_samp * state_size + cudakernel_id;
            const int64_t index_bg = i_samp * channel_size * height * width + i_channel * height * width + y_inp_bg * width + x_inp_bg;
            const int64_t index_k = i_samp * channel_size * height * width + i_channel * height * width + y_inp_k * width + x_inp_k;
            auto grad_out_val = *(grad_output + index_out);
            auto inp_bg_val = *(input + index_bg);
            auto inp_k_val = *(input + index_k);
            auto grad_inp_bg = grad_input + index_bg;
            auto grad_inp_k = grad_input + index_k;
            
            atomicAdd(grad_inp_bg, grad_out_val * inp_k_val);
            atomicAdd(grad_inp_k, grad_out_val * inp_bg_val);
        }
}

std::vector<at::Tensor> lacorr2d_backward_cuda(
    at::Tensor input,
    at::Tensor grad_output,
    int kernel_height,
    int kernel_width,
    int stride_height,
    int stride_width) {
    const int batch_size = input.size(0);
    const int channel_size = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    AT_ASSERT(kernel_width <= width, "kernel_width must be lesser than or equal to width")
    AT_ASSERT(kernel_height <= height, "kernel_height must be lesser than or equal to height")
    AT_ASSERT(stride_width <= width, "stride_width must be lesser than or equal to width")
    AT_ASSERT(stride_height <= height, "stride_height must be lesser than or equal to height")

    const int n_corr_w = (width - kernel_width) / stride_width + 1;
    const int n_corr_h = (height - kernel_height) / stride_height + 1;
    const int n_corr = n_corr_w * n_corr_h;
    const int corr_size = kernel_height * kernel_width;

    const int64_t state_size = (int64_t)channel_size * n_corr * corr_size;

    auto grad_input = at::zeros_like(input);

    const int threads = 1024;
    const dim3 blocks((state_size + threads - 1) / threads, corr_size, batch_size);
    AT_DISPATCH_FLOATING_TYPES(input.type(), "lacorr2d_forward_cuda", ([&] {
        lacorr2d_backward_cuda_kernel<scalar_t><<<blocks, threads>>>(
            input.data<scalar_t>(),
            grad_output.data<scalar_t>(),
            grad_input.data<scalar_t>(),
            kernel_height,
            kernel_width,
            stride_height,
            stride_width,
            n_corr_h,
            n_corr_w,
            channel_size,
            height,
            width,
            state_size);
    }));
    return {grad_input};
}